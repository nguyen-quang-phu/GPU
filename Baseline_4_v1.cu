#include <stdio.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <algorithm> 
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// Sequential radix sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
void sortByThrust(const uint32_t * in, int n, 
    uint32_t * out)
{
    thrust::device_vector<uint32_t> dv_out(in, in + n);
	thrust::sort(dv_out.begin(), dv_out.end());
	thrust::copy(dv_out.begin(), dv_out.end(), out);
}
void sortByDevice(const uint32_t * in, int n, 
    uint32_t * out, 
    int nBits, int  blockSize)
{
    int nBins = 1 << nBits; // số bin
    int m=(n - 1) / blockSize + 1;// gridSize

    // cấp phát
    // local hist
    int **localHist = (int **)malloc(m * sizeof(int *)); 
    for (int i=0; i<m; i++)
    {
        localHist[i] = (int *)malloc(nBins * sizeof(int)); 
    }

    // scan
    int **scan = (int **)malloc(m * sizeof(int *)); 
    for (int i=0; i<m; i++)
    {
        scan[i] = (int *)malloc(nBins * sizeof(int)); 
    }

    // chỉ số bắt đầu
    int **start = (int **)malloc(m * sizeof(int *)); 
    for (int i=0; i<m; i++)
    {
        start[i] = (int *)malloc(nBins * sizeof(int)); 
    }

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;




    for (int bit = 0;  bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        // gán localHist=0
        for (int i=0; i<m; i++)
        {
            memset(localHist[i], 0, nBins * sizeof(int));
        }

        // Tính localHist
        for(int blockIdx=0;blockIdx<m;blockIdx++)
        {
            for(int threadIdx=0;threadIdx<blockSize;threadIdx++)
            {
                int i=blockSize*blockIdx+threadIdx;
                if(i<n)
                {
                    int bin = (src[i] >> bit) & (nBins - 1);
                    localHist[blockIdx][bin]++;
                }
            }
        }

        // cấp phát scan=0
        for (int i=0; i<m; i++)
        {
            memset(scan[i], 0, nBins * sizeof(int));
        }

        // tính exclusive scan
        for(int bin=0; bin<nBins;bin++)
        {
            for (int blockIdx=0;blockIdx<m;blockIdx++)
            {
                if(blockIdx==0&&bin==0)
                {
                    scan[blockIdx][bin]=0;
                }
                else
                {
                    if (blockIdx==0)
                    {
                        scan[blockIdx][bin]=scan[m-1][bin-1]+localHist[m-1][bin-1];
                    }
                    else
                    {
                        scan[blockIdx][bin]=scan[blockIdx-1][bin]+localHist[blockIdx-1][bin];
                    }
                }
            }
        }

        
        // sắp xếp cục bộ
        for(int blockIdx=0;blockIdx<m;blockIdx++)
        {
            for(int index=0;index<blockSize-1;index++)
            {
                for(int threadIdx=0;threadIdx<blockSize-1-index;threadIdx++)
                {
                    int i=blockIdx*blockSize+threadIdx;
                    if(i+1<n)
                    {
                        if(((src[i] >> bit) & (nBins - 1))>((src[i+1] >> bit) & (nBins - 1)))
                        {
                            uint32_t temp=src[i];
                            src[i]=src[i+1];
                            src[i+1]=temp;
                        }
                    }
                    
                }
            }
        }

        // cấp phát start=-1
        for (int i=0; i<m; i++)
        {
            memset(start[i], -1, nBins * sizeof(int));
        }

        // tính chỉ số bắt đầu
        for(int blockIdx=0;blockIdx<m;blockIdx++)
        {
            for(int threadIdx=0;threadIdx<blockSize;threadIdx++)
            {
                int i=blockIdx*blockSize+threadIdx;
                if (i<n)
                {
                    if(threadIdx==0)
                    {
                        start[blockIdx][((src[i] >> bit) & (nBins - 1))]=threadIdx;
                    }
                    else
                    {
                        if(((src[i] >> bit) & (nBins - 1))!=((src[i-1] >> bit) & (nBins - 1)))
                        {
                            start[blockIdx][((src[i] >> bit) & (nBins - 1))]=threadIdx;
                        }
                    }
                }
            }
        }

        //scatter
        for(int blockIdx=0;blockIdx<m;blockIdx++)
        {
            for(int threadIdx=0;threadIdx<blockSize;threadIdx++)
            {
                int i=blockIdx*blockSize+threadIdx;
                if(i<n)
                {
                    int bin = (src[i] >> bit) & (nBins - 1);
                    int rank=scan[blockIdx][bin]+threadIdx-start[blockIdx][bin];
                    dst[rank]=src[i];
                }
            }
        }
        uint32_t * temp = src;
        src = dst;
        dst = temp; 
    }
    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    for (int i=0; i<m; i++)
    {
        free(localHist[i]);
    }
    free(localHist);

    for (int i=0; i<m; i++)
    {
        free(scan[i]);
    }
    free(scan);
    for (int i=0; i<m; i++)
    {
        free(start[i]);
    }
    free(start); 
    free(originalSrc);
}
// Radix sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits,
        bool useDevice=false, int  blockSizes=512)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix sort by Thrust\n");
        sortByThrust(in, n, out);
    }
    else // use device
    {
    	printf("\nRadix sort by device\n");
        sortByDevice(in, n, out, nBits, blockSizes);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 24) + 1;
    n = 65536;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
        in[i] = rand();
    //printArray(in, n);

    // SET UP NBITS
    int nBits = 4; // Default
    if (argc > 1)
        nBits = atoi(argv[1]);
    printf("\nNum bits per digit: %d\n", nBits);

    // DETERMINE BLOCK SIZES
    int blockSizes=512; // One for histogram, one for scan
    if (argc == 3)
    {
        blockSizes = atoi(argv[2]);
    }
    printf("\block size: %d", blockSizes);

    // SORT BY HOST
    sort(in, n, correctOut, nBits);
    // printArray(correctOut, n);
    
    // SORT BY DEVICE
    sort(in, n, out, nBits, true, blockSizes);
    // printArray(out,n);
    checkCorrectness(out, correctOut, n);
    
    // FREE MEMORIES 
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
