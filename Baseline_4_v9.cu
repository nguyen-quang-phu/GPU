#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <algorithm> 
#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}


struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};


// Thrust Radix Sort
void sortByThrust(const uint32_t * in, int n, 
    uint32_t * out)
{
    thrust::device_vector<uint32_t> dv_out(in, in + n);
	thrust::sort(dv_out.begin(), dv_out.end());
	thrust::copy(dv_out.begin(), dv_out.end(), out);
}


__global__ void computeLocalHist(uint32_t * in, int n, uint32_t * scan, int nBins, int bit)
{
    extern __shared__ int s_hist[];
    int i=blockDim.x*blockIdx.x+threadIdx.x;
    for(int stride=0;stride<nBins;stride+=blockDim.x)
        if(threadIdx.x+stride<nBins)
            s_hist[threadIdx.x+stride]=0;
    __syncthreads();

    if(i<n)
    {
        int bin=(in[i]>>bit)&(nBins-1);// lấy nBits ra để tính xem phần tử này thuộc bin nào
        atomicAdd(&s_hist[bin], 1);
    }
    __syncthreads();// syncthreads để chắc chắn các phần tử trong block đã được tính trong s_hist

    for(int stride=0;stride<nBins;stride+=blockDim.x)
        if(threadIdx.x+stride<nBins)
            scan[(threadIdx.x+stride)*gridDim.x+blockIdx.x]=s_hist[threadIdx.x+stride];
            // hist[nBins*blockIdx.x+threadIdx.x+stride]=s_hist[threadIdx.x+stride];
}


__global__ void scanBlkKernel(uint32_t * in, int n, uint32_t * out, uint32_t * blkSums)
{   
    extern __shared__ uint32_t value[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        value[threadIdx.x] = in[i];
    }
    
    for (unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
        __syncthreads();
        int tmp;
        if (threadIdx.x < n - stride)
            tmp = value[threadIdx.x-stride];
        else
            tmp = 0;
        __syncthreads();
        value[threadIdx.x] += tmp;
    }
    
    blkSums[blockIdx.x] = value[blockDim.x - 1];
    __syncthreads();
    if (i<n) {
        if(threadIdx.x==0)
        {
            out[i]=0;
        } 
        else
        out[i]=value[threadIdx.x-1];
    }
}



__global__ void addSumScan(uint32_t * out, int n, uint32_t * blkSums)
{   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && blockIdx.x > 0) 
    {
        out[i] = out[i] + blkSums[blockIdx.x - 1];
    }
}



__global__ void radixSort1bit(uint32_t * in, int n, uint32_t * out,int nBits, int bit,int nBins, uint32_t* starts)
{   int i = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ uint32_t value[];
    __shared__ uint32_t start[256];

    for(int indexbit=0;indexbit<nBits;indexbit++)
    {
        if (i < n) 
        {
            value[threadIdx.x] = ((((in[i] >> bit) & (nBins - 1)) >> indexbit) & 1);
        }
        __syncthreads();
        for(int stride=1;stride<blockDim.x;stride*=2)
        {
            int temp=0;
            if(threadIdx.x>=stride)
            {
                temp=value[threadIdx.x-stride];// lấy phần tử trước đó stride bước
            }
            __syncthreads();// chắc chắn giá trị năm trước stride bước đã được lấy vào bộ nhớ thanh ghi
            if(threadIdx.x>=stride )
            {
                value[threadIdx.x]+=temp;
            }
            __syncthreads();// chắc chắn các giá trị đã được cộng xong
        }
        int nZeros=0;
        if(blockIdx.x*blockDim.x+blockDim.x<=n)
            nZeros = blockDim.x - value[blockDim.x-2] -((((in[blockIdx.x*blockDim.x+blockDim.x-1] >> bit) & (nBins - 1)) >> indexbit) & 1);
        else
        {
            if(n%blockDim.x>=2)
            nZeros = n%blockDim.x - value[n%blockDim.x-2] - ((((in[n-1] >> bit) & (nBins - 1)) >> indexbit) & 1);
            else
            nZeros = n%blockDim.x  - ((((in[n-1] >> bit) & (nBins - 1)) >> indexbit) & 1);
        }
        if (i<n)
        {
            if(threadIdx.x==0)
            {
                if (((((in[i] >> bit) & (nBins - 1)) >> indexbit) & 1)==0)
                {
                    out[i]=in[i];
                }
                else
                    out[nZeros+blockIdx.x*blockDim.x]=in[i];
            }
            else
            {
                if(((((in[i] >> bit) & (nBins - 1)) >> indexbit) & 1)==0)
                {
                    out[i-value[threadIdx.x-1]]=in[i];
                }
                else
                {
                    out[nZeros+value[threadIdx.x-1]+blockIdx.x*blockDim.x]=in[i];
                }
            }
        }
        __syncthreads();
        uint32_t *tmp=in;
        in=out;
        out=tmp;
    }
    if (i<n)
    {
        if(threadIdx.x==0)
        {
            start[((in[i] >> bit) & (nBins - 1))]=threadIdx.x;
        }
        else
        {
            if(((in[i] >> bit) & (nBins - 1))!=((in[i-1] >> bit) & (nBins - 1)))
            {
                start[((in[i] >> bit) & (nBins - 1))]=threadIdx.x;
                starts[blockIdx.x*nBins+((in[i] >> bit) & (nBins - 1))]=start[((in[i] >> bit) & (nBins - 1))];
            }
        }
    }    
}


__global__ void scatter(uint32_t * in, int n, uint32_t * out,int nBits, int bit,int nBins, uint32_t* start, uint32_t* histScan)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        int bin = (in[i] >> bit) & (nBins - 1);
        int rank=histScan[bin*gridDim.x+blockIdx.x]+threadIdx.x-start[nBins*blockIdx.x+bin];
        out[rank]=in[i];
    }
}


void sortByDevice(const uint32_t * in, int n, 
    uint32_t * out, 
    int nBits, int  blockSizes)
{
    int nBins = 1 << nBits; // số bin
    int m = (n - 1) / blockSizes + 1;// gridSize
    dim3 blockSize(blockSizes);
    dim3 blockSizeScan(blockSizes);

    dim3 gridSize((n - 1) / blockSize.x + 1);
    dim3 gridSizeScan((nBins*m - 1) / blockSizeScan.x + 1);
    // cấp phát
   
    // scan
    uint32_t *d_scan, *d_blkSums, *d_histScan, *d_blkOuts, *d_starts;
    uint32_t *histScan = (uint32_t *)malloc(m*nBins * sizeof(uint32_t));
    uint32_t *blkSums = (uint32_t *)malloc(m*nBins*sizeof(uint32_t));
    uint32_t* starts1D=(uint32_t *) malloc(m*nBins*sizeof(uint32_t));

    CHECK(hipMalloc(&d_scan, nBins*m * sizeof(uint32_t)));
    CHECK(hipMalloc(&d_blkSums,gridSizeScan.x*sizeof(uint32_t)));
    CHECK(hipMalloc(&d_blkOuts,m*nBins*sizeof(uint32_t)));
    CHECK(hipMalloc(&d_starts,m*nBins*sizeof(uint32_t)));
    CHECK(hipMalloc(&d_histScan,m*nBins*sizeof(uint32_t)));
    // chỉ số bắt đầu

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    uint32_t * d_in,*d_out, *d_tmp;
    CHECK(hipMalloc(&d_in,n * sizeof(uint32_t)));
    CHECK(hipMalloc(&d_out,n * sizeof(uint32_t)));
    CHECK(hipMalloc(&d_tmp,n * sizeof(uint32_t)));

    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    size_t bytes = gridSizeScan.x * sizeof(uint32_t);
    uint32_t * in_tmp = (uint32_t *)malloc(bytes);
    uint32_t * out_tmp = (uint32_t*)malloc(bytes);
    CHECK(hipMemcpy(d_in, src, n * sizeof(uint32_t), hipMemcpyHostToDevice));

    GpuTimer timerTmp1,timerTmp2,timerTmp3,timerTmp4,timerTmp5; 
    float time1,time2,time3,time4,time5;
    time1=time2=time3=time4=time5=0;


    for (int bit = 0;  bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        // Tính local hist bỏ vào d_scan
        timerTmp1.Start();
        computeLocalHist<<<gridSize, blockSize, blockSizes*sizeof(uint32_t)>>>(d_in, n, d_scan, nBins,bit);
        timerTmp1.Stop();
        time1 = time1 + timerTmp1.Elapsed();
        timerTmp2.Start(); 

        // // Tính exclusive scan bỏ vào d_histscan
        scanBlkKernel<<<gridSizeScan,blockSizeScan,blockSizes*sizeof(uint32_t)>>>(d_scan,m*nBins,d_histScan,d_blkSums);
        CHECK(hipMemcpy(in_tmp, d_blkSums, gridSizeScan.x * sizeof(uint32_t), hipMemcpyDeviceToHost));
        out_tmp[0] = in_tmp[0];
	    for (int i = 1; i < gridSizeScan.x; i++)
	    {
	    	out_tmp[i] = out_tmp[i - 1] + in_tmp[i];
	    }
		CHECK(hipMemcpy(d_blkOuts, out_tmp, gridSizeScan.x * sizeof(uint32_t), hipMemcpyHostToDevice));
        addSumScan<<<gridSizeScan,blockSizeScan>>>(d_histScan, n, d_blkOuts);
    	hipDeviceSynchronize();
		CHECK(hipGetLastError());

        timerTmp2.Stop();
        time2 = time2 + timerTmp2.Elapsed();
        timerTmp3.Start();

        // Radix Sort 1 bit
        radixSort1bit<<<gridSize,blockSize,blockSizes*sizeof(uint32_t)>>>(d_in,n,d_out,nBits,bit,nBins, d_starts);

        timerTmp3.Stop();
        time3 = time3 + timerTmp3.Elapsed();
        timerTmp5.Start();

        // Scatter
        scatter<<<gridSize,blockSize,blockSizes*sizeof(uint32_t)>>>(d_in,n,d_out,nBits,bit,nBins,d_starts,d_histScan);

        timerTmp5.Stop();
        time5 = time5 + timerTmp5.Elapsed();

        d_tmp = d_in;
        d_in = d_out;
        d_out = d_tmp;
    }
    printf("Time (local hist): %.3f ms\n", time1);
    printf("Time (exclusive scan): %.3f ms\n", time2);
    printf("Time (local sort) + Time (start value): %.3f ms\n", time3);
    printf("Time (scatter): %.3f ms\n", time5);

    
    CHECK(hipMemcpy(src, d_in, n * sizeof(uint32_t), hipMemcpyDeviceToHost));
    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    hipFree(d_scan);
    hipFree(d_blkSums);
    hipFree(d_histScan);
    hipFree(d_in);
    hipFree(d_out);
    free(originalSrc);
}
// Radix sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits,
        bool useDevice=false, int  blockSizes=512)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix sort by Thrust\n");
        sortByThrust(in, n, out);
    }
    else // use device
    {
    	printf("\nRadix sort by device\n");
        sortByDevice(in, n, out, nBits, blockSizes);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            // printf("%i %i",out[i+100] , correctOut[i+100]);
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < 100; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 24) + 1;
    // n = 1000000;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
        in[i] = rand();
    //printArray(in, n);

    // SET UP NBITS
    int nBits = 4; // Default
    if (argc > 1)
        nBits = atoi(argv[1]);
    printf("\nNum bits per digit: %d\n", nBits);

    // DETERMINE BLOCK SIZES
    int blockSizes=512; // One for histogram, one for scan
    if (argc == 3)
    {
        blockSizes = atoi(argv[2]);
    }
    printf("\nblock size: %d", blockSizes);

    // SORT BY HOST
    sort(in, n, correctOut, nBits);
    // printArray(correctOut, n);
    
    // SORT BY DEVICE
    sort(in, n, out, nBits, true, blockSizes);
    // printArray(out,n);
    checkCorrectness(out, correctOut, n);
    
    // FREE MEMORIES 
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}